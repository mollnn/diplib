#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

template <typename T>
__global__ void __kernel_ImgAlgInterp_interpBilinear_cuda(T *dest_ptr_d, T *src_ptr_d, float *coords_d, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int dest_x = blockDim.x * blockIdx.x + threadIdx.x;
    int dest_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dest_x >= dest_width || dest_y >= dest_height)
        return;

    int dest_idx = dest_y * dest_width + dest_x;

    float x = coords_d[2 * dest_idx + 0];
    float y = coords_d[2 * dest_idx + 1];

    int x0 = x;
    int x1 = x0 + 1;
    int y0 = y;
    int y1 = y0 + 1;

    bool valid_x0 = 0 <= x0 && x0 < src_width;
    bool valid_x1 = 0 <= x1 && x1 < src_width;
    bool valid_y0 = 0 <= y0 && y0 < src_height;
    bool valid_y1 = 0 <= y1 && y1 < src_height;

    int idx00 = y0 * src_width + x0;
    int idx01 = y1 * src_width + x0;
    int idx10 = y0 * src_width + x1;
    int idx11 = y1 * src_width + x1;

    T val00 = valid_x0 && valid_y0 ? src_ptr_d[idx00] : default_value;
    T val01 = valid_x0 && valid_y1 ? src_ptr_d[idx01] : default_value;
    T val10 = valid_x1 && valid_y0 ? src_ptr_d[idx10] : default_value;
    T val11 = valid_x1 && valid_y1 ? src_ptr_d[idx11] : default_value;

    float v = ((x1 - x) * val00 + (x - x0) * val10) * (y1 - y) +
              ((x1 - x) * val01 + (x - x0) * val11) * (y - y0);

    dest_ptr_d[dest_y * dest_width + dest_x] = v;
}

template <typename T>
void  __ImgAlgInterp_interpBilinear_cuda_(T *dest_ptr, T *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int src_size = src_width * src_height;
    int dest_size = dest_width * dest_height;

    T *dest_ptr_d;
    T *src_ptr_d;
    float *coords_d;

    hipMalloc(&dest_ptr_d, dest_size * sizeof(T));
    hipMalloc(&src_ptr_d, src_size * sizeof(T));
    hipMalloc(&coords_d, dest_size * 2 * sizeof(float));

    hipMemcpy(src_ptr_d, src_ptr, src_size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(coords_d, coords, dest_size * 2 * sizeof(float), hipMemcpyHostToDevice);

    int block_width = 8;
    int block_height = 8;

    int grid_width = (dest_width + block_width - 1) / block_width;
    int grid_height = (dest_height + block_height - 1) / block_height;

    __kernel_ImgAlgInterp_interpBilinear_cuda<T><<<dim3(grid_width, grid_height), dim3(block_width, block_height)>>>(dest_ptr_d, src_ptr_d, coords_d, dest_width, dest_height, src_width, src_height, default_value);
    hipDeviceSynchronize();

    hipMemcpy(dest_ptr, dest_ptr_d, dest_size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(dest_ptr_d);
    hipFree(src_ptr_d);
    hipFree(coords_d);
}

// template
// void  __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda <uint8_t>(uint8_t *dest_ptr, uint8_t *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, uint8_t default_value);

// template
// void  __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda <uint16_t>(T *dest_ptr, T *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, T default_value);

// template
// void  __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda <float>(T *dest_ptr, T *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, T default_value);


extern "C" void __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda_epi8(uint8_t *dest_ptr, uint8_t *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, uint8_t default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<uint8_t>(dest_ptr, src_ptr, coords, dest_width,dest_height,src_width,src_height,default_value);
}

extern "C" void __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda_epi16(uint16_t *dest_ptr, uint16_t *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, uint16_t default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<uint16_t>(dest_ptr, src_ptr, coords, dest_width,dest_height,src_width,src_height,default_value);
}

extern "C" void __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda_ps(float *dest_ptr, float *src_ptr, float *coords, int dest_width, int dest_height, int src_width, int src_height, float default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<float>(dest_ptr, src_ptr, coords, dest_width,dest_height,src_width,src_height,default_value);
}

extern "C" void __declspec(dllexport) test()
{
    printf("ok");
}

// int main()
// {
//     int w = 10000;
//     int h = 10000;
//     uint16_t *a = (uint16_t *)malloc(w * h * sizeof(uint16_t));
//     uint16_t *b = (uint16_t *)malloc(w * h * sizeof(uint16_t));
//     float *c = (float *)malloc(w * h * 2 * sizeof(float));
//     for (int i = 0; i < w * h; i++)
//     {
//         c[i * 2 + 0] = i * i * 1e-4;
//         c[i * 2 + 1] = i * i * 1e-4;
//     }
//     __ImgAlgInterp_interpBilinear_cuda<uint16_t>(b, a, c, w, h, w, h, 0);
// }