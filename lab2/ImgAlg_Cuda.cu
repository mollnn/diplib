#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

template <typename T>
__global__ void __kernel_ImgAlgInterp_interpBilinear_cuda(T *dest_ptr_d, T *src_ptr_d, float *x_coords_d, float *y_coords_d, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int dest_x = blockDim.x * blockIdx.x + threadIdx.x;
    int dest_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dest_x >= dest_width || dest_y >= dest_height)
        return;

    int dest_idx = dest_y * dest_width + dest_x;

    float x = x_coords_d[dest_idx];
    float y = y_coords_d[dest_idx];

    int x0 = x;
    int x1 = x0 + 1;
    int y0 = y;
    int y1 = y0 + 1;

    bool valid_x0 = 0 <= x0 && x0 < src_width;
    bool valid_x1 = 0 <= x1 && x1 < src_width;
    bool valid_y0 = 0 <= y0 && y0 < src_height;
    bool valid_y1 = 0 <= y1 && y1 < src_height;

    int idx00 = y0 * src_width + x0;
    int idx01 = y1 * src_width + x0;
    int idx10 = y0 * src_width + x1;
    int idx11 = y1 * src_width + x1;

    T val00 = valid_x0 && valid_y0 ? src_ptr_d[idx00] : default_value;
    T val01 = valid_x0 && valid_y1 ? src_ptr_d[idx01] : default_value;
    T val10 = valid_x1 && valid_y0 ? src_ptr_d[idx10] : default_value;
    T val11 = valid_x1 && valid_y1 ? src_ptr_d[idx11] : default_value;

    float v = ((x1 - x) * val00 + (x - x0) * val10) * (y1 - y) +
              ((x1 - x) * val01 + (x - x0) * val11) * (y - y0);

    dest_ptr_d[dest_y * dest_width + dest_x] = v;
}

template <typename T>
void __ImgAlgInterp_interpBilinear_cuda_(T *dest_ptr, T *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int src_size = src_width * src_height;
    int dest_size = dest_width * dest_height;

    T *dest_ptr_d;
    T *src_ptr_d;
    float *x_coords_d;
    float *y_coords_d;

    hipMalloc(&dest_ptr_d, dest_size * sizeof(T));
    hipMalloc(&src_ptr_d, src_size * sizeof(T));
    hipMalloc(&x_coords_d, dest_size * sizeof(float));
    hipMalloc(&y_coords_d, dest_size * sizeof(float));

    hipMemcpy(src_ptr_d, src_ptr, src_size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(x_coords_d, x_coords, dest_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_coords_d, y_coords, dest_size * sizeof(float), hipMemcpyHostToDevice);

    int block_width = 8;
    int block_height = 8;

    int grid_width = (dest_width + block_width - 1) / block_width;
    int grid_height = (dest_height + block_height - 1) / block_height;

    __kernel_ImgAlgInterp_interpBilinear_cuda<T><<<dim3(grid_width, grid_height), dim3(block_width, block_height)>>>(dest_ptr_d, src_ptr_d, x_coords_d, y_coords_d, dest_width, dest_height, src_width, src_height, default_value);
    hipDeviceSynchronize();

    hipMemcpy(dest_ptr, dest_ptr_d, dest_size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(dest_ptr_d);
    hipFree(src_ptr_d);
    hipFree(x_coords_d);
    hipFree(y_coords_d);
}

extern "C" void __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda_epi8(uint8_t *dest_ptr, uint8_t *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, uint8_t default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<uint8_t>(dest_ptr, src_ptr, x_coords, y_coords, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda_epi16(uint16_t *dest_ptr, uint16_t *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, uint16_t default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<uint16_t>(dest_ptr, src_ptr, x_coords, y_coords, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __declspec(dllexport) __ImgAlgInterp_interpBilinear_cuda_ps(float *dest_ptr, float *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, float default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<float>(dest_ptr, src_ptr, x_coords, y_coords, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __declspec(dllexport) test()
{
    printf("ok");
}
