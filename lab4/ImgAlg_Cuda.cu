#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define DLL_EXP
#include "ImgAlg_Cuda.h"

/////////////////////////////////////////////////////
// ImgAlgInterp
/////////////////////////////////////////////////////

template <typename T>
__device__ T ___device_ImgAlgInterp_pxInterpBilinear_cuda_(float x, float y, T *src_ptr_d, int src_width, int src_height, T default_value)
{
    int x0 = x;
    int x1 = x0 + 1;
    int y0 = y;
    int y1 = y0 + 1;

    bool valid_x0 = 0 <= x0 && x0 < src_width;
    bool valid_x1 = 0 <= x1 && x1 < src_width;
    bool valid_y0 = 0 <= y0 && y0 < src_height;
    bool valid_y1 = 0 <= y1 && y1 < src_height;

    int idx00 = y0 * src_width + x0;
    int idx01 = y1 * src_width + x0;
    int idx10 = y0 * src_width + x1;
    int idx11 = y1 * src_width + x1;

    T val00 = valid_x0 && valid_y0 ? src_ptr_d[idx00] : default_value;
    T val01 = valid_x0 && valid_y1 ? src_ptr_d[idx01] : default_value;
    T val10 = valid_x1 && valid_y0 ? src_ptr_d[idx10] : default_value;
    T val11 = valid_x1 && valid_y1 ? src_ptr_d[idx11] : default_value;

    float v = ((x1 - x) * val00 + (x - x0) * val10) * (y1 - y) +
              ((x1 - x) * val01 + (x - x0) * val11) * (y - y0);

    return v;
}

template <typename T>
__global__ void __kernel_ImgAlgInterp_interpBilinear_cuda(T *dest_ptr_d, T *src_ptr_d, float *x_coords_d, float *y_coords_d, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int dest_x = blockDim.x * blockIdx.x + threadIdx.x;
    int dest_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dest_x >= dest_width || dest_y >= dest_height)
        return;

    int dest_idx = dest_y * dest_width + dest_x;

    float x = x_coords_d[dest_idx];
    float y = y_coords_d[dest_idx];

    dest_ptr_d[dest_y * dest_width + dest_x] = ___device_ImgAlgInterp_pxInterpBilinear_cuda_(x, y, src_ptr_d, src_width, src_height, default_value);
}

template <typename T>
void __ImgAlgInterp_interpBilinear_cuda_(T *dest_ptr, T *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int src_size = src_width * src_height;
    int dest_size = dest_width * dest_height;

    T *dest_ptr_d;
    T *src_ptr_d;
    float *x_coords_d;
    float *y_coords_d;

    hipMalloc(&dest_ptr_d, dest_size * sizeof(T));
    hipMalloc(&src_ptr_d, src_size * sizeof(T));
    hipMalloc(&x_coords_d, dest_size * sizeof(float));
    hipMalloc(&y_coords_d, dest_size * sizeof(float));

    hipMemcpy(src_ptr_d, src_ptr, src_size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(x_coords_d, x_coords, dest_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_coords_d, y_coords, dest_size * sizeof(float), hipMemcpyHostToDevice);

    int block_width = 16;
    int block_height = 16;

    int grid_width = (dest_width + block_width - 1) / block_width;
    int grid_height = (dest_height + block_height - 1) / block_height;

    __kernel_ImgAlgInterp_interpBilinear_cuda<T><<<dim3(grid_width, grid_height), dim3(block_width, block_height)>>>(dest_ptr_d, src_ptr_d, x_coords_d, y_coords_d, dest_width, dest_height, src_width, src_height, default_value);
    hipDeviceSynchronize();

    hipMemcpy(dest_ptr, dest_ptr_d, dest_size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(dest_ptr_d);
    hipFree(src_ptr_d);
    hipFree(x_coords_d);
    hipFree(y_coords_d);
}

////////////////////////////////////////
// ImgAlgAffine
////////////////////////////////////////

__device__ void ___device_ImgAlgAffine_pxAffineTransformCoordEvaluate_cuda(float *x_coords_dest_d, float *y_coords_dest_d, int coords_idx, float *mat_d, float x, float y)
{
    float x1 = mat_d[0] * x + mat_d[1] * y + mat_d[2];
    float y1 = mat_d[3] * x + mat_d[4] * y + mat_d[5];
    x_coords_dest_d[coords_idx] = x1;
    y_coords_dest_d[coords_idx] = y1;
}

template <typename T>
__global__ void __kernel_ImgAlgAffine_affineTransform_cuda(T *dest_ptr_d, T *src_ptr_d, float *mat_d, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    int dest_x = blockDim.x * blockIdx.x + threadIdx.x;
    int dest_y = blockDim.y * blockIdx.y + threadIdx.y;
    float src_x;
    float src_y;
    ___device_ImgAlgAffine_pxAffineTransformCoordEvaluate_cuda(&src_x, &src_y, 0, mat_d, dest_x, dest_y);
    // dest_ptr_d[dest_y * dest_width + dest_x] = ___device_ImgAlgInterp_pxInterpBilinear_cuda_<T>(src_x, src_y, src_ptr_d, src_width, src_height, default_value);
    dest_ptr_d[dest_y * dest_width + dest_x] = 127;
    dest_ptr_d[0] = 127;
    dest_ptr_d[1] = 127;
    dest_ptr_d[2] = 127;
}

template <typename T>
void __ImgAlgAffine_affineTransform_cuda_(T *dest_ptr, T *src_ptr, float *mat, int dest_width, int dest_height, int src_width, int src_height, T default_value)
{
    mat[8] = 111;
    int src_size = src_width * src_height;
    int dest_size = dest_width * dest_height;

    T *dest_ptr_d;
    T *src_ptr_d;
    float *mat_d;

    hipMalloc(&dest_ptr_d, dest_size * sizeof(T));
    hipMalloc(&src_ptr_d, src_size * sizeof(T));
    hipMalloc(&mat_d, 9 * sizeof(float));

    hipMemcpy(src_ptr_d, src_ptr, src_size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(mat_d, mat, 9 * sizeof(float), hipMemcpyHostToDevice);

    int block_width = 16;
    int block_height = 16;

    int grid_width = (dest_width + block_width - 1) / block_width;
    int grid_height = (dest_height + block_height - 1) / block_height;

    __kernel_ImgAlgAffine_affineTransform_cuda<T><<<dim3(grid_width, grid_height), dim3(block_width, block_height)>>>(dest_ptr_d, src_ptr_d, mat_d, dest_width, dest_height, src_width, src_height, default_value);
    hipDeviceSynchronize();

    hipMemcpy(dest_ptr, dest_ptr_d, dest_size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(dest_ptr_d);
    hipFree(src_ptr_d);
    hipFree(mat_d);
}

template <typename T>
__global__ void __kernel_Img_conv2d_cuda(T *dest_ptr_d, T *src_ptr_d, float *kernel_d, int dest_width, int dest_height, int src_width, int src_height, int kernel_width, int kernel_height, T default_value)
{
    int dest_x = blockDim.x * blockIdx.x + threadIdx.x;
    int dest_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (!(dest_x >= 0 && dest_y >= 0 && dest_x < dest_width && dest_y < dest_height))
        return;

    int kernel_width_half = kernel_width / 2;
    int kernel_height_half = kernel_height / 2;

    float local_sum = 0;

    for (int i = -kernel_height_half; i <= kernel_height_half; i++)
    {
        for (int j = -kernel_width_half; j <= kernel_width_half; j++)
        {
            int kernel_x = j + kernel_width_half;
            int kernel_y = i + kernel_height_half;
            int src_x = dest_x - j;
            int src_y = dest_y - i;

            float kernel_value = kernel_d[kernel_y * kernel_width + kernel_x];

            // 分支导致了线程束分化（但是开销比 padding 小）
            if (src_x >= 0 && src_y >= 0 && src_x < src_width && src_y < src_height)
            {
                local_sum += src_ptr_d[src_y * src_width + src_x] * kernel_value;
            }
            else
            {
                local_sum += default_value * kernel_value;
            }
        }
    }

    dest_ptr_d[dest_y * dest_width + dest_x] = local_sum;
}

template <typename T>
void __Img_conv2d_cuda_(T *dest_ptr, T *src_ptr, float *kernel, int dest_width, int dest_height, int src_width, int src_height, int kernel_width, int kernel_height, T default_value)
{
    int src_size = src_width * src_height;
    int dest_size = dest_width * dest_height;
    int kernel_size = kernel_width * kernel_height;

    T *dest_ptr_d;
    T *src_ptr_d;
    float *kernel_d;

    hipMalloc(&dest_ptr_d, dest_size * sizeof(T));
    hipMalloc(&src_ptr_d, src_size * sizeof(T));
    hipMalloc(&kernel_d, kernel_size * sizeof(float));

    hipMemcpy(src_ptr_d, src_ptr, src_size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(kernel_d, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    int block_width = 16;
    int block_height = 16;

    int grid_width = (dest_width + block_width - 1) / block_width;
    int grid_height = (dest_height + block_height - 1) / block_height;

    __kernel_Img_conv2d_cuda<T><<<dim3(grid_width, grid_height), dim3(block_width, block_height)>>>(dest_ptr_d, src_ptr_d, kernel_d, dest_width, dest_height, src_width, src_height, kernel_width, kernel_height, default_value);
    hipDeviceSynchronize();

    hipMemcpy(dest_ptr, dest_ptr_d, dest_size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(dest_ptr_d);
    hipFree(src_ptr_d);
    hipFree(kernel_d);
}

//////////////////////////////////////////
//// Specifications (for export)
//// MinGW 与 MSVC 的 C++ ABI 不兼容，故采用 C 风格导出
//////////////////////////////////////////

extern "C" void __ImgAlgInterp_interpBilinear_cuda_epu8(uint8_t *dest_ptr, uint8_t *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, uint8_t default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<uint8_t>(dest_ptr, src_ptr, x_coords, y_coords, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __ImgAlgInterp_interpBilinear_cuda_epu16(uint16_t *dest_ptr, uint16_t *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, uint16_t default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<uint16_t>(dest_ptr, src_ptr, x_coords, y_coords, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __ImgAlgInterp_interpBilinear_cuda_ps(float *dest_ptr, float *src_ptr, float *x_coords, float *y_coords, int dest_width, int dest_height, int src_width, int src_height, float default_value)
{
    __ImgAlgInterp_interpBilinear_cuda_<float>(dest_ptr, src_ptr, x_coords, y_coords, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __ImgAlgAffine_affineTransform_cuda_epu8(uint8_t *dest_ptr, uint8_t *src_ptr, float *mat, int dest_width, int dest_height, int src_width, int src_height, uint8_t default_value)
{
    __ImgAlgAffine_affineTransform_cuda_<uint8_t>(dest_ptr, src_ptr, mat, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __ImgAlgAffine_affineTransform_cuda_epu16(uint16_t *dest_ptr, uint16_t *src_ptr, float *mat, int dest_width, int dest_height, int src_width, int src_height, uint16_t default_value)
{
    __ImgAlgAffine_affineTransform_cuda_<uint16_t>(dest_ptr, src_ptr, mat, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __ImgAlgAffine_affineTransform_cuda_ps(float *dest_ptr, float *src_ptr, float *mat, int dest_width, int dest_height, int src_width, int src_height, float default_value)
{
    __ImgAlgAffine_affineTransform_cuda_<float>(dest_ptr, src_ptr, mat, dest_width, dest_height, src_width, src_height, default_value);
}

extern "C" void __Img_conv2d_cuda_epu8(uint8_t *dest_ptr, uint8_t *src_ptr, float *kernel, int dest_width, int dest_height, int src_width, int src_height, int kernel_width, int kernel_height, uint8_t default_value)
{
    __Img_conv2d_cuda_<uint8_t>(dest_ptr, src_ptr, kernel, dest_width, dest_height, src_width, src_height, kernel_width, kernel_height, default_value);
}

extern "C" void __Img_conv2d_cuda_epu16(uint16_t *dest_ptr, uint16_t *src_ptr, float *kernel, int dest_width, int dest_height, int src_width, int src_height, int kernel_width, int kernel_height, uint16_t default_value)
{
    __Img_conv2d_cuda_<uint16_t>(dest_ptr, src_ptr, kernel, dest_width, dest_height, src_width, src_height, kernel_width, kernel_height, default_value);
}

extern "C" void __Img_conv2d_cuda_ps(float *dest_ptr, float *src_ptr, float *kernel, int dest_width, int dest_height, int src_width, int src_height, int kernel_width, int kernel_height, float default_value)
{
    __Img_conv2d_cuda_<float>(dest_ptr, src_ptr, kernel, dest_width, dest_height, src_width, src_height, kernel_width, kernel_height, default_value);
}
